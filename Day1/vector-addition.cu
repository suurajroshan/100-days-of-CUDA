// code from NVIDIA's CUDA C/C++ course

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void initWith(float num, float *a, int N){
    int i;
    for (i=0; i<N; ++i){
        a[i] = num;
    }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N){
    int i, idxWithinGrid, gridStride;
    idxWithinGrid = threadIdx.x + blockIdx.x * blockDim.x;
    gridStride = gridDim.x * blockDim.x;
    for (i=idxWithinGrid; i<N; i+=gridStride){
        result[i] = a[i]+b[i];
    }
}

void CheckElementsAre(float target, float *array, int N){
    int i;
    for (i=0; i<N;++i){
        if (array[i] != target){
            printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
            exit(1);
        }
    }
    printf("SUCCESS! All values added correctly.\n");
}

int main(){
    const int N = 2<<20;
    size_t size = N*sizeof(float);

    float *a;
    float *b;
    float *c;

    checkCuda( hipMallocManaged(&a, size) );
    checkCuda( hipMallocManaged(&b, size) );
    checkCuda( hipMallocManaged(&c, size) );

    initWith(4, a, N);
    initWith(5, b, N);
    initWith(0, c, N);

    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 256;
    numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);
    checkCuda( hipGetLastError() );
    checkCuda( hipDeviceSynchronize() );

    CheckElementsAre(9, c, N);

    checkCuda( hipFree(a) );
    checkCuda( hipFree(b) );
    checkCuda( hipFree(c) );

}