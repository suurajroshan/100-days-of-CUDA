// Code for naive matrix-matrix multiplication
// Code from NVIDIA's CUDA course


#include <stdio.h>
#include "naiveMatMulGPU.h"

#define N 1024

/*
 * This CPU function already works, and will run to create a solution matrix
 * against which to verify your work building out the matrixMulGPU kernel.
 */

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}

int main()
{
  int *a, *b, *c_cpu, *c_gpu; // Allocate a solution matrix for both the CPU and the GPU operations

  int size = N * N * sizeof (int); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  // Initialize memory; create 2D matrices
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
    }

  /*
   * Assign `threads_per_block` and `number_of_blocks` 2D values
   * that can be used in matrixMulGPU above.
   */

  int threads = 4;
  dim3 threads_per_block(threads, threads);
  dim3 number_of_blocks((N + threads_per_block.x - 1) / threads_per_block.x, 
                          (N + threads_per_block.y - 1) / threads_per_block.y);

  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu, N );
  hipDeviceSynchronize();

  // Call the CPU version to check our work
  matrixMulCPU( a, b, c_cpu );

  // Compare the two answers to make sure they are equal
  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  // If success then time the kernel
  int nIter = 300;
  hipEvent_t NaiveStart, NaiveStop;
  hipEventCreate(&NaiveStart); hipEventCreate(&NaiveStop);
  hipEventRecord(NaiveStart);
  for (int n = 0; n < nIter; ++n){
    matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu, N );
  }
  hipEventRecord(NaiveStop); hipEventSynchronize(NaiveStop);
  float msecTotal = 0.0f;
  hipEventElapsedTime(&msecTotal, NaiveStart, NaiveStop);
  float msecPerMatrixMul = msecTotal / nIter;
  printf("%f\n",msecPerMatrixMul);
  // Free all our allocated memory
  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
}
